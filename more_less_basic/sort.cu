
#include <hip/hip_runtime.h>

#include <stdio.h>

#define arraySize 5
#define threadPerBlock 5
// Kernel Function for Rank sort
__global__ void addKernel(int *d_a, int *d_b) {
  int count = 0;
  int tid = threadIdx.x;
  int ttid = blockIdx.x * threadPerBlock + tid;
  int val = d_a[ttid];
  __shared__ int cache[threadPerBlock];
  
  for (int i = tid; i < arraySize; i += threadPerBlock) {
    cache[tid] = d_a[i];
    __syncthreads();
    for (int j = 0; j < threadPerBlock; ++j)
      if (val > cache[j])
        count++;
    __syncthreads();
  }
  d_b[count] = val;
}

int main() {
  // Define Host and Device Array
  int h_a[arraySize] = {5, 9, 3, 4, 8};
  int h_b[arraySize];
  int *d_a, *d_b;

  // Allocate Memory on the device
  hipMalloc((void **)&d_b, arraySize * sizeof(int));
  hipMalloc((void **)&d_a, arraySize * sizeof(int));

  // Copy input vector from host memory to device memory.
  hipMemcpy(d_a, h_a, arraySize * sizeof(int), hipMemcpyHostToDevice);

  // Launch a kernel on the GPU with one thread for each element.
  addKernel<<<arraySize / threadPerBlock, threadPerBlock>>>(d_a, d_b);

  // Wait for device to finish operations
  hipDeviceSynchronize();
  // Copy output vector from GPU buffer to host memory.
  hipMemcpy(h_b, d_b, arraySize * sizeof(int), hipMemcpyDeviceToHost);
  printf("The Enumeration sorted Array is: \n");
  for (int i = 0; i < arraySize; i++) {
    printf("%d\n", h_b[i]);
  }
  // Free up device memory
  hipFree(d_a);
  hipFree(d_b);
  return 0;
}