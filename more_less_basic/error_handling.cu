#include "stdio.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

void handle_errors(void){
    int *d_a;
    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**)&d_a, sizeof(int));
    printf("Status: %d, hipSuccess: %d\n", cudaStatus, hipSuccess);

    int *h_a;
    cudaStatus = hipMemcpy(d_a,&h_a, sizeof(int), hipMemcpyHostToDevice);
    printf("Status: %d, hipSuccess: %d\n", cudaStatus, hipSuccess);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(d_a);
        }

    cudaStatus = hipGetLastError();
    printf("Status: %d, hipSuccess: %d\n", cudaStatus, hipSuccess);
    
    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
}

void properties(void){
    hipDeviceProp_t device_Property;
    hipGetDeviceProperties(&device_Property, 0);
    printf("Max per procesor: %d\n", device_Property.maxThreadsPerMultiProcessor);
    printf("Max per block: %d\n", device_Property.maxThreadsPerBlock);
    printf("CUDA stream: %d\n", device_Property.deviceOverlap);
}

int main(void){
    handle_errors();
    properties();
    

    return 0;
}