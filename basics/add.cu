#include <iostream>
#include <hip/hip_runtime.h>



__global__ void gpuAdd(int *d_a, int *d_b, int *d_c) 
{
   *d_c = *d_a + *d_b;
}

int main() 
{
   int h_a, h_b, h_c;
   int *d_a, *d_b, *d_c;

   h_a = 1;
   h_b = 4;

   hipMalloc((void**)&d_a, sizeof(int));
   hipMalloc((void**)&d_b, sizeof(int));
   hipMalloc((void**)&d_c, sizeof(int));

   hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

   gpuAdd<<<1,1>>>(d_a, d_b, d_c);

   hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
   printf("Passing Parameter by Reference Output: %d + %d = %d\n", h_a, h_b, h_c);

   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);
   return 0;
}