
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void gpu_shared_memory(float *d_a) {
  int i, index = threadIdx.x;
  float average, sum = 0.0f;
  // Defining shared memory
  __shared__ float sh_arr[10];

  sh_arr[index] = d_a[index];
  // This directive ensure all the writes to shared memory have completed

  __syncthreads();
  for (i = 0; i <= index; i++) {
    sum += sh_arr[i];
  }
  average = sum / (index + 1.0f);
  d_a[index] = average;

  // This statement is redundant and will have no effect on overall code
  // execution
  sh_arr[index] = average;
}

int main(int argc, char **argv) {
  float h_a[10];
  float *d_a;

  // Initialize host Array
  for (int i = 0; i < 10; i++) {
    h_a[i] = i;
  }

  // allocate global memory on the device
  hipMalloc((void **)&d_a, sizeof(float) * 10);

  // copy data from host memory  to device memory
  hipMemcpy((void *)d_a, (void *)h_a, sizeof(float) * 10,
             hipMemcpyHostToDevice);
  gpu_shared_memory<<<1, 10>>>(d_a);

  // copy the modified array back to the host
  hipMemcpy((void *)h_a, (void *)d_a, sizeof(float) * 10,
             hipMemcpyDeviceToHost);
  printf("Use of Shared Memory on GPU: \n");

  for (int i = 0; i < 10; i++) {
    printf("The running average after %d element is %f \n", i, h_a[i]);
  }
  return 0;
}