#include "stdio.h"
#include <hip/hip_runtime.h>

#include <iostream>

// Defining two constants
__constant__ int constant_f;
__constant__ int constant_g;
#define N 5

// Kernel function for using constant memory
__global__ void gpu_constant_memory(float *d_in, float *d_out) {
  // Getting thread index for current kernel
  int tid = threadIdx.x;
  d_out[tid] = constant_f * d_in[tid] + constant_g;
}

int main(void) {
  // Defining Arrays for host
  float h_in[N], h_out[N];
  // Defining Pointers for device
  float *d_in, *d_out;

  int h_f = 2;
  int h_g = 20;

  // allocate the memory on the cpu
  hipMalloc((void **)&d_in, N * sizeof(float));
  hipMalloc((void **)&d_out, N * sizeof(float));

  // Initializing Array
  for (int i = 0; i < N; i++) {
    h_in[i] = i;
  }

  // Copy Array from host to device
  hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
  // Copy constants to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(constant_f), &h_f, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(constant_g), &h_g, sizeof(int));

  // Calling kernel with one block and N threads per block
  gpu_constant_memory<<<1, N>>>(d_in, d_out);

  // Coping result back to host from device memory
  hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

  // Printing result on console
  printf("Use of Constant memory on GPU \n");
  for (int i = 0; i < N; i++) {
    printf("The expression for index %f is %f\n", h_in[i], h_out[i]);
  }

  hipFree(d_in);
  hipFree(d_out);
  return 0;
}