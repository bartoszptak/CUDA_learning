#include "hip/hip_runtime.h"
#include "stdio.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define NUM_THREADS 10
#define N 10

// Define texture reference for 1-d access
texture<float, 1, hipReadModeElementType> textureRef;

__global__ void gpu_texture_memory(int n, float *d_out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    float temp = tex1D(textureRef, float(idx));
    d_out[idx] = temp;
  }
}

int main() {
  // Calculate number of blocks to launch
  int num_blocks = N / NUM_THREADS + ((N % NUM_THREADS) ? 1 : 0);
  float *d_out;
  // allocate space on the device for the results
  hipMalloc((void **)&d_out, sizeof(float) * N);
  // allocate space on the host for the results
  float *h_out = (float *)malloc(sizeof(float) * N);
  float h_in[N];
  for (int i = 0; i < N; i++) {
    h_in[i] = float(i);
  }
  // Define CUDA Array
  hipArray *cu_Array;
  hipMallocArray(&cu_Array, &textureRef.channelDesc, N, 1);

  hipMemcpyToArray(cu_Array, 0, 0, h_in, sizeof(float) * N,
                    hipMemcpyHostToDevice);

  // bind a texture to the CUDA array
  hipBindTextureToArray(textureRef, cu_Array);

  gpu_texture_memory<<<num_blocks, NUM_THREADS>>>(N, d_out);

  // copy result to host
  hipMemcpy(h_out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
  printf("Use of Texture memory on GPU: \n");
  // Print the result
  for (int i = 0; i < N; i++) {
    printf("Average between two nearest element is : %f\n", h_out[i]);
  }
  free(h_out);
  hipFree(d_out);
  hipFreeArray(cu_Array);
  hipUnbindTexture(textureRef);
}